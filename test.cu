
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<fstream>
#include<string>
#include<sstream>
#include<ctime>

#define MAX_M 500000
using namespace std;

#define CUDA_CALL(x) do { hipError_t err=(x); \
	if(err!=hipSuccess) { \
	printf("Error %s at %s: %d",hipGetErrorString(err),__FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)  

__device__  int getProcessCount(const int nodes){
	/*
	In case the number of threads is less than the 
	total number of nodes in the graph. In this case,
	each thread handles more than one node, and the
	exact number is given by this function
	*/
	int no_threads = gridDim.x*blockDim.x;
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if (tid>=nodes)
		return 0;
	else if (tid< nodes % no_threads)
		return (nodes+no_threads-1)/no_threads;
	else
		return nodes/no_threads;

}


__global__ void fixMatching(int* cmatch, int* rmatch,int *nodes)
{
	/*
	To handle any race conditions that may have arisen.
	We don't explicitly prevent the race conditions, rather
	opting to fix them at each iteration. But each iteration
	guarantees at least one augmenting path hence the number of
	iterations is bounded
	*/
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int process_cnt = getProcessCount(*nodes);
	for(int i=0;i<process_cnt;i++){
		int col_vertex=i*(gridDim.x*blockDim.x)+tid;
		//Race condition
		if(cmatch[rmatch[col_vertex]]!=col_vertex)
			rmatch[col_vertex]=-1;
	}

}

__global__ void initBfsArray(int* bfs_array, int* amatch,int* nodes){
	/*
	Kernel to initialize the BFS array.
	Sets the node to -1 if already matched, 
	and to 0 if the node has not been matched yet
	*/
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int process_cnt = getProcessCount(*nodes);
	for(int i=0;i<process_cnt;i++){
		int col_vertex=i*(gridDim.x*blockDim.x)+tid;
		if(amatch[col_vertex]>-1)
			bfs_array[col_vertex]=-1;
		else if(amatch[col_vertex]==-1)
			bfs_array[col_vertex]=0;
	}
}



__global__ void bfs_edges(int* predecessor, int* bfs_level,int* bfs_array,int* xadj,int* adj,int *nodes, 
		int* rmatch, bool* vertex_inserted, bool* augmenting_path_found, int col_vertex,int start_index)
{
	/*
	Kernel called from the GPU for dynamic parallelism
	Not many changes from the bfs kernel
	*/	   
	int j = threadIdx.x+start_index;
	int neighbour_row=adj[j];
	int col_match=rmatch[neighbour_row];
	if(col_match>-1)
	{
		if(bfs_array[col_match]==-1)
		{
			*vertex_inserted=true;
			bfs_array[col_match]=*bfs_level+1;
			predecessor[neighbour_row]=col_vertex;
		}
	}
	else
	{
		if(col_match==-1)
		{
			rmatch[neighbour_row]=-2;
			predecessor[neighbour_row]=col_vertex;
			*augmenting_path_found=true;
		}
	}
}

__global__ void bfs(int* predecessor, int* bfs_level,int* bfs_array,int* xadj,int* adj,int *nodes, 
		int* rmatch, bool* vertex_inserted, bool* augmenting_path_found)
{
	/*
	Main kernel. Iterates through all the edges
	of a particular node.
	*/
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int process_cnt = getProcessCount(*nodes);
	for(int i=0;i<process_cnt;i++)
	{
		int col_vertex=i*(gridDim.x*blockDim.x)+tid;
		if(bfs_array[col_vertex]==*bfs_level)
		{
			int threads = xadj[col_vertex+1]-xadj[col_vertex];
			//If the node has positive number of neighbours
			if (threads>0)
				bfs_edges<<<1,threads,0>>>( predecessor,bfs_level, bfs_array, xadj, adj,nodes, rmatch,vertex_inserted,augmenting_path_found, col_vertex,xadj[col_vertex]);
		}
	}
}

__global__ void alternate(int* cmatch, int* rmatch, int* nodes,int* predecessor)
{
	/*
	If an augmenting path ending at the vertex 
	has been found, iterate through the predecessor 
	array ie. traverse the augmenting path and alterate 
	the edges to augment it.
	*/
	int tid  = blockIdx.x *blockDim.x+threadIdx.x;
	int process_vent=getProcessCount(*nodes);
	for(int i=0;i<process_vent;i++)
	{
		int row_vertex=i*(gridDim.x*blockDim.x)+tid;
		if(rmatch[row_vertex]==-2)
		{
			while(row_vertex!=-1)
			{
				int matched_col=predecessor[row_vertex];
				int matched_row=cmatch[matched_col];
				if (matched_row!=-1)
					if(predecessor[matched_row]==matched_col)
						break;
				cmatch[matched_col]=row_vertex;
				rmatch[row_vertex]=matched_col;
				row_vertex=matched_row;
			}
		}
	}
}



int main(){

	int *d_predecessor,*d_bfs_level,*d_bfs_array, *d_xadj, *d_adj, *d_amatch,*d_bmatch;

	int nodes= 6511;
    	int edges= 0;
	int readnode[2];
	int n=0;
	int i=0;
	
	int* d_nodes;	
	hipMalloc(&d_nodes,sizeof(int));
	hipMemcpy(d_nodes,&nodes,sizeof(int),hipMemcpyHostToDevice);


	int p=0;
	int index=0;
	int flag1=0;

	// Various timers	
	hipEvent_t start, stop,kernel_start,kernel_stop;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	float all_time=0 ,kernel_time = 0;
	hipEventRecord(start,0);	

	//Adjacency list only of first part of bipartite graph	
	int *xadj=(int*)malloc(nodes*sizeof(int));	
	int *adj =(int*)malloc(MAX_M*sizeof(int));
	
	//CPT Time for File IO
	clock_t cpub = clock();
	
	string line;
    	ifstream myfile ("data4.txt");
	/*
	The file should be sorted numerically. use "sort -n"
	1 2
	3 4 
	is valid
	
	3 4
	1 2 
	is not valid
	*/
	
	nodes = nodes+1;
	/*
	FILE IO and conversion into compact adjacency list

	One workaround done here is the increment of node by 1
	for the file io. This is basically adding a phantom node
	and removing it after. The data input file should also have 
	the line 
	<nodes+1> <nodes+1>
	at the end. This was to make the file io and conversion easier.
	The issue if this line doesn't exist can be explained with an
	example. If nodes is 1000 and nodes 999, 998 do not have edges,
	ie they do not turn up in the file, then they will not have
	a legal value set for their xadj and adj.
	*/
	if (myfile.is_open()){
		p=0;
		index=0;
		flag1=0;
		while ( getline (myfile,line) ){
			edges++;
			stringstream S;
		       	S<<line;
			i=0;
			while( S >> n ) {
				readnode[i]=n;
				i++;
			}
			if(!p)
				xadj[p]=0;
			while(p!=readnode[0]-1)
			{
				p++;
				xadj[p]=index;
				if(p>(nodes-1))
				{
					flag1=1;
					break;
				}
			}
			if(flag1)
				break;
			adj[index]=readnode[1]-1;
			index++;

		}
		myfile.close();
	}
	clock_t cpue = clock();
	double cpu_time = 1000* double(cpue - cpub) / CLOCKS_PER_SEC;
	printf("FILE IO Time :%f ms\n",cpu_time);             	
	
	//Removal of the phantom node and edge we added
	nodes = nodes-1;
	edges = edges-1;	

	int *amatch =(int*)malloc(nodes*sizeof(int));	
	int *bmatch =(int*)malloc(nodes*sizeof(int));
	int *bfs_array =(int*)malloc(nodes*sizeof(int));
    
	memset(amatch,-1,sizeof(int)*nodes);
	memset(bmatch,-1,sizeof(int)*nodes);
	
	int bfs_level = 0;
	

	hipMalloc(&d_amatch,sizeof(int)*nodes);
	hipMalloc(&d_bmatch,sizeof(int)*nodes);
	hipMalloc(&d_predecessor,sizeof(int)*nodes);
	hipMalloc(&d_bfs_level,sizeof(int)*nodes);
	hipMalloc(&d_bfs_array,sizeof(int)*nodes);
	hipMalloc(&d_xadj,sizeof(int)*(nodes+1));
	hipMalloc(&d_adj,sizeof(int)*edges);

	hipEventRecord(kernel_start,0);	

	//Note the nodes + 1 here. This is so that the final edge doesn't
	//access illegal memory	
	hipMemcpy(d_xadj, xadj, sizeof(int)*(nodes+1),hipMemcpyHostToDevice);
	hipMemcpy(d_adj,adj, sizeof(int)*edges,hipMemcpyHostToDevice);

	//Number of nodes a thread should handle
	//Default is one
	int nops = 1;
	dim3 threads(4);
	dim3 blocks((nodes+threads.x-1)/(threads.x*nops));

	bool* d_augmenting_path_found;
	bool* d_vertex_inserted;
	bool* augmenting_path_found = (bool*)malloc(sizeof(bool));
	bool* vertex_inserted = (bool*)malloc(sizeof(bool));
	
     	*augmenting_path_found = true;


	hipMalloc(&d_augmenting_path_found,sizeof(bool));
     	hipMalloc(&d_vertex_inserted,sizeof(bool));
		
	hipMemset(d_amatch,-1,sizeof(int)*nodes);
	hipMemset(d_bmatch,-1,sizeof(int)*nodes);
	
	while (*augmenting_path_found){
	/*
	Main loop. While either an augmenting path was found or
	a new vertex was inserted into the set of vertices for
	consideration.
	
	The program flow mainly involves the three kernels, apart
	from the one used to initialize. Kernel bfs() is the most work
	intensive one. alternate() is only relevant if an augmenting 
	path is found. fixMatching() is a support kernel for fixing
	errors due to race conditions
	*/
		initBfsArray<<<blocks,threads>>> (d_bfs_array,d_amatch,d_nodes);
		
		*vertex_inserted= true;		
		hipMemset(d_bfs_level,0,sizeof(int));  
		bfs_level = 0;

		while (*vertex_inserted){	
			//Reset flags
			hipMemset(d_vertex_inserted,false,sizeof(bool));  
			hipMemset(d_augmenting_path_found,false,sizeof(bool));  
			
			bfs<<<blocks,threads>>> (d_predecessor,d_bfs_level,d_bfs_array,d_xadj,d_adj,d_nodes,d_bmatch,d_vertex_inserted,d_augmenting_path_found);
			
			hipMemcpy(augmenting_path_found,d_augmenting_path_found,sizeof(bool),hipMemcpyDeviceToHost);
			hipMemcpy(vertex_inserted,d_vertex_inserted,sizeof(bool),hipMemcpyDeviceToHost);

			if (*augmenting_path_found){
				break;
			}
			bfs_level+=1;
			hipMemcpy(d_bfs_level,&bfs_level,sizeof(int),hipMemcpyHostToDevice);
		}
		alternate<<<blocks,threads>>> (d_amatch,d_bmatch,d_nodes,d_predecessor);
		fixMatching<<<blocks,threads>>> (d_amatch,d_bmatch,d_nodes);
	}

	hipMemcpy(amatch,d_amatch, sizeof(int)*nodes,hipMemcpyDeviceToHost);	
	hipMemcpy(bmatch,d_bmatch, sizeof(int)*nodes,hipMemcpyDeviceToHost);	


	hipEventRecord(kernel_stop,0);
	hipEventSynchronize(kernel_stop);
	hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);

	//Final check of race conditions
	int maxmat=0;
	for(int i =0;i<nodes;++i){
		if(amatch[i]!=-1)
			if(bmatch[amatch[i]]==i)
				maxmat++;
	}
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&all_time, start, stop);
	
	printf("\nOverall Time : %f ms\nKernel Time :%f ms\n",all_time,kernel_time);
	cout << "Size of maximum matching is "<<maxmat<<endl; 
	cout << "Minimum size of fleet required is "<<(nodes-maxmat)<<endl; 
	free(xadj);
	free(adj);
}

